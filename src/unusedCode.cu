#include "hip/hip_runtime.h"
// save some currently unused functions to integrate back into the code later on

__global__ void generate2DRings(DeviceParticlesType particles)
{

    const float R = 0.38;
    const float r = 0.3;
    const float seperationX = 1;
    const float seperationY = 0;
    const float speed = 0.5;

    const float ringSize = particles.size()/2;
    const float a = M_PI * (R*R-r*r);
    const float ringMass = rho0 * a;
    const float particleMass = ringMass/ringSize;

    // find the starting index
    int startingIndex = (r*r) * ringSize;
    int lastIteration=0;
    while(abs(startingIndex-lastIteration)>5)
    {
        lastIteration = startingIndex;
        startingIndex = ((r/R)*(r/R)) * (ringSize+startingIndex);
    }

    // calculate the particle distance
    f2_t posA;
    f2_t posB;
    float l = R * sqrt(10/(ringSize+startingIndex));
    float theta = 2 * sqrt(M_PIf32*10);
    posA.x = l * cos(theta);
    posA.y = l * sin(theta);
    l = R * sqrt(11/(ringSize+startingIndex));
    theta = 2 * sqrt(M_PIf32*11);
    posB.x = l * cos(theta);
    posB.y = l * sin(theta);
    f1_t spacing = length(posA-posB);
    printf("particle seperation: %f\n ", spacing);

    INIT_EACH(particles, MPU_COMMA_LIST(POS,MASS,VEL,DENSITY),
              {
                      thrust::random::default_random_engine rng;
                      rng.discard(i);
                      thrust::random::uniform_real_distribution<float> dist(-0.1f*spacing,0.1f*spacing);

                      float index;
                      if(i<ringSize)
                      {
                          index = i + startingIndex;
                          pi.pos.x = seperationX/2;
                          pi.pos.y = seperationY/2;
                          pi.vel.x = -speed/2;
                      }
                      else
                      {
                          index = i-ringSize + startingIndex;
                          pi.pos.x = -seperationX/2;
                          pi.pos.y = -seperationY/2;
                          pi.vel.x = speed/2;
                      }

                      l = R * sqrt(index/(ringSize+startingIndex));
                      theta = 2 * sqrt(M_PIf32*index);
                      pi.pos.x += l * cos(theta);
                      pi.pos.y += l * sin(theta);

                      pi.mass = particleMass;
                      pi.density = rho0;
              });
}

__global__ void generateSquares(DeviceParticlesType particles)
{
    INIT_EACH(particles, MPU_COMMA_LIST(POS,MASS,VEL,DENSITY),
              {
                      float ratio = 0.99f;
                      float spacing = H/3;
                      printf("spacing: %f\n",spacing);
                      int squareSize1 = particles.size() * ratio;
                      int squareSize2 = particles.size() * (1.0f-ratio);
                      int sideres1 = sqrt(float(squareSize1));
                      int sideres2 = sqrt(float(squareSize2));
                      float side1 = (sideres1-1) * spacing;
                      float side2 = (sideres2-1) * spacing;

                      const float a1 = side1*side1;
                      const float a2 = side2*side2;
                      const float squareMass1 = rho0 * a1;
                      const float squareMass2 = rho0 * a2;
                      const float particleMass1 = squareMass1/squareSize1;
                      const float particleMass2 = squareMass2/squareSize2;

                      const float speed = 5;
                      const float seperationX = 1;
                      const float seperationY = 0;

                      thrust::random::default_random_engine rng;
                      rng.discard(i);
                      thrust::random::uniform_real_distribution<float> dist(-0.1f*spacing,0.1f*spacing);

                      if(i < squareSize1)
                      {
                          pi.pos.x = -side1 / 2 + (i%sideres1) *spacing + dist(rng);
                          pi.pos.y = -side1 / 2 + (i/sideres1) *spacing + dist(rng);
                          pi.pos.x -= seperationX/2;
                          pi.pos.y -= seperationY/2;
                          pi.vel.x = speed * (1-ratio);
                          pi.mass = particleMass1;
                      }
                      else
                      {
                          pi.pos.x = -side2 / 2 + ((i-squareSize1)%sideres2) *spacing + dist(rng);
                          pi.pos.y = -side2 / 2 + ((i-squareSize1)/sideres2) *spacing + dist(rng);
                          pi.pos.x += seperationX/2;
                          pi.pos.y += seperationY/2;
                          pi.vel.x = -speed * ratio;
                          pi.mass = particleMass2;
                      }
                      pi.density = rho0;
              })
}

__global__ void generateRect(DeviceParticlesType particles)
{
    INIT_EACH(particles, MPU_COMMA_LIST(POS,MASS,VEL,DENSITY),
              {
                      int rows = 150;
                      float offsetY = -0.45;

                      float spacing = H/3;
                      printf("spacing: %f\n",spacing);

                      int coatCols = 2.0f/spacing;
                      int coatSize = coatCols * 3;

                      int size = particles.size() - coatSize;
                      int cols = size / rows;
                      float side1 = (cols-1) * spacing;
                      float side2 = (rows-1) * spacing;
                      const float a = side1*side2;
                      const float mass = rho0 * a;
                      const float particleMass = mass/size;

                      thrust::random::default_random_engine rng;
                      rng.discard(i);
                      thrust::random::uniform_real_distribution<float> dist(-0.1f*spacing,0.1f*spacing);

                      if(i < coatSize)
                      {
                          pi.pos.x = -1.0f + ((i) % coatCols) * spacing + dist(rng);
                          pi.pos.y = -0.99f + ((i) / coatCols) * spacing + dist(rng);
                      }
                      else
                      {
                          pi.pos.x = -side1 / 2 + ((i-coatSize) % cols) * spacing + dist(rng);
                          pi.pos.y = -side2 / 2 + ((i-coatSize) / cols) * spacing + dist(rng);
                          pi.pos.y += offsetY;
                      }
                      pi.mass = particleMass;
                      pi.density = rho0;
              })
}

__global__ void generate2DHydroNBSystem(DeviceParticlesType particles)
{
    INIT_EACH(particles, MPU_COMMA_LIST(POS,MASS,VEL,DENSITY),
              {
                      thrust::random::default_random_engine rng;
                      rng.discard(i*particles.size());
                      thrust::random::uniform_real_distribution<float> dist(-1.0f,1.0f);

                      do
                      {
                          pi.pos.x = dist(rng);
                          rng.discard(particles.size());
                          pi.pos.y = dist(rng);
                          rng.discard(particles.size());
                          pi.pos.z = dist(rng);
                          rng.discard(particles.size());
                      }
                      while(length(pi.pos) > 1);

                      pi.mass = mass;
                      pi.density = rho0;

                      pi.vel.x = pi.pos.y * rsqrtf(3);
                      pi.vel.y = - pi.pos.x * rsqrtf(3);
              });
}

__global__ void computeDensity(DeviceParticlesType particles)
{
    DO_FOR_EACH_PAIR_SM( BLOCK_SIZE, particles, MPU_COMMA_LIST(SHARED_POSM),
                         MPU_COMMA_LIST(POS,MASS,DENSITY),
                         MPU_COMMA_LIST(POS,MASS), MPU_COMMA_LIST(DENSITY),
                         MPU_COMMA_LIST(POS,MASS),
                         {},
                         {
                                 const f3_t rij = pi.pos-pj.pos;
                                 const f1_t r2 = dot(rij,rij);
                                 f1_t r = sqrt(r2);
                                 if(r<=H)
                                 {
                                     pi.density += pj.mass * kernel::Wspline<dimension>(r,H);
                                 }
                         },
                         {})
}

__global__ void window2DBound(DeviceParticlesType particles)
{
    DO_FOR_EACH(particles, MPU_COMMA_LIST(POS,VEL),
                MPU_COMMA_LIST(POS,VEL),
                MPU_COMMA_LIST(POS,VEL),
                {
                        if(pi.pos.x > 1)
                        {
                            pi.pos.x=1;
                            pi.vel.x -= 1.5*pi.vel.x;
                        }
                        else if(pi.pos.x < -1)
                        {
                            pi.pos.x=-1;
                            pi.vel.x -= 1.5*pi.vel.x;
                        }
                        if(pi.pos.y > 1)
                        {
                            pi.pos.y=1;
                            pi.vel.y -= 1.5*pi.vel.y;
                        }
                        else if(pi.pos.y < -1)
                        {
                            pi.pos.y=-1;
                            pi.vel.y = 0;
                            pi.vel.x = 0;

                        }
                })
}

__global__ void integrate(DeviceParticlesType particles, f1_t dt)
{
    DO_FOR_EACH(particles, MPU_COMMA_LIST(POS,VEL,ACC,XVEL,DENSITY,DENSITY_DT,DSTRESS,DSTRESS_DT),
                MPU_COMMA_LIST(POS,VEL,ACC,XVEL,DENSITY,DENSITY_DT,DSTRESS,DSTRESS_DT),
                MPU_COMMA_LIST(POS,VEL,DENSITY,DSTRESS),
                {
                        // eqn of motion
                        pi.vel += pi.acc * dt;
                        pi.pos += (pi.vel+0.6f*pi.xvel) * dt;

                        // density
                        pi.density += pi.density_dt * dt;

                        if(pi.density < 0)
                        pi.density = 0;

                        // deviatoric stress
                        pi.dstress += pi.dstress_dt * dt;

                        plasticity(pi.dstress,mohrCoulombYieldStress(tan(friction_angle),eos::murnaghan(pi.density,rho0, BULK, dBULKdP),cohesion));
                })
}

__global__ void generate2DNBSystem(DeviceParticlesType particles)
{
    INIT_EACH(particles, MPU_COMMA_LIST(POS,MASS,VEL),
              {
                      thrust::random::default_random_engine rng;
                      rng.discard(i);
                      thrust::random::uniform_real_distribution<float> dist(-1.0f,1.0f);

                      pi.pos.x = dist(rng);
                      pi.pos.y = dist(rng);
                      pi.pos.z = 0.0f;
                      pi.mass = 1.0f/particles.size();

                      pi.vel = cross(pi.pos,{0.0f,0.0f, 0.75f});
              });
}

__global__ void nbodyForces(DeviceParticlesType particles, f1_t eps2)
{
    DO_FOR_EACH_PAIR_SM( BLOCK_SIZE, particles, SHARED_POSM, MPU_COMMA_LIST(POS,MASS,VEL,ACC),
                         MPU_COMMA_LIST(POS,MASS,VEL), MPU_COMMA_LIST(ACC), MPU_COMMA_LIST(POS, MASS), {},
                         {
                                 f3_t r = pi.pos - pj.pos;
                                 f1_t distSqr = dot(r, r) + eps2;
                                 f1_t invDist = rsqrt(distSqr);
                                 f1_t invDistCube = invDist * invDist * invDist;
                                 pi.acc -= r * pj.mass * invDistCube;
                         },
                         {
                                 pi.acc -= pi.vel * 0.01;
                         })
}


//-------------------------------------------------------------------
// create HostParticleBuffer in a save way

namespace detail {
template<typename Tuple>
struct mhpb_impl;

template<typename ... TArgs>
struct mhpb_impl<std::tuple<TArgs...>>
{
template<typename ...ConstrArgs>
static auto make_hpb(ConstrArgs &&... args)
{
    return HostParticleBuffer<TArgs...>(std::forward<ConstrArgs>(args)...);
}
};
}

/**
 * @brief Creates a HostParticleBuffer in a save way from a std::tuple of attributes, making sure all attributes are in the correct order.
 * @tparam TupleType the tuple to create the particle from
 * @return the created particle
 */
template <typename TupleType, typename ...ConstrArgs, std::enable_if_t< mpu::is_instantiation_of_v<std::tuple,TupleType> , int> _null =0>
auto make_hpb(ConstrArgs && ... args)
{
    return detail::mhpb_impl< reorderd_t<TupleType,host_base_order >>::make_hpb(std::forward<ConstrArgs>(args)...);
}

/**
 * @brief Creates a HostParticleBuffer in a save way from another HostParticleBuffer type (like what you get from concatenating particles), making sure all attributes are in the correct order.
 * @tparam BufferType the particle to create
 * @return the created particle
 */
template <typename BufferType, typename ...ConstrArgs, std::enable_if_t< mpu::is_instantiation_of_v<HostParticleBuffer,BufferType> , int> _null =0>
auto make_hpb(ConstrArgs && ... args)
{
    return detail::mhpb_impl< reorderd_t<particle_to_tuple_t<BufferType> ,host_base_order>>::make_hpb(std::forward<ConstrArgs>(args)...);
}

/**
 * @brief Creates a HostParticleBuffer in a save way from a list of attributes, making sure all attributes are in the correct order.
 * @tparam TypeArgs particle attributes to sort and use
 * @return the created particle
 */
template <typename ...TypeArgs, typename ...ConstrArgs, std::enable_if_t< (sizeof...(TypeArgs)>1), int> _null =0>
auto make_hpb(ConstrArgs && ... args)
{
    return make_hpb<std::tuple<TypeArgs...>>(std::forward<ConstrArgs>(args)...);
}

/**
 * @brief The type of HostParticleBuffer generated when calling make_hpb with the template arguments Args
 */
template <typename ...Args>
using make_hpb_t = decltype(make_hpb<Args...>());

//-------------------------------------------------------------------
// merge multiple particles

/**
 * @brief Merge multiple HostParticleBuffer. A new HostParticleBuffer with all attributes from all input particles is created and values are copied.
 *          If input particles share an attribute the value from particle Pa or the last particle with the attribute in question is used.
 * @param pa the first particle
 * @param pb the second particle
 * @return a new particle with all attributes from pa and pb
 */
template <typename PTa, typename ...PTs>
auto merge_hpb(const PTa& pa,const PTs& ... ps)
{
    auto p = make_hpb< particle_concat_t<PTa, PTs...> >();

    int t[] = {0, ((void)(p=ps),1)...};
    (void)t[0]; // silence compiler warning about t being unused
    p = pa;

    return p;
}

/**
 * @brief The type of HostParticleBuffer generated when calling merge_hpb with the template arguments Args
 */
template <typename ... Args>
using merge_hpb_t = decltype(merge_particles<Args...>(Args()...));
