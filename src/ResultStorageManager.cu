/*
 * GraSPH2
 * ResultStorageManager.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the ResultStorageManager class
 *
 * Copyright (c) 2018 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "ResultStorageManager.h"
//--------------------


// function definitions of the ResultStorageManager class
//-------------------------------------------------------------------

ResultStorageManager::ResultStorageManager(std::string directory, std::string prefix, int maxJobs)
    : m_directory(directory), m_prefix(prefix), m_terminateWorker(false), m_maxQueue(maxJobs), m_numberJobs(0)
{
    assert_critical(m_maxQueue>1, "ResultStorageManager", "Can't work with Maximum job number below 1.")

    m_startTime = mpu::timestamp("%Y-%m-%d_%H:%M");
    m_workerThread = std::thread(&ResultStorageManager::worker, this);
}

void ResultStorageManager::worker()
{
    std::unique_lock<std::mutex> lck(m_workerMutex);
    while(!m_terminateWorker)
    {
        // wait until there is work to do
        m_workSignal.wait(lck);

        while(m_numberJobs > 0)
        {
            // download data from gpu to cpu if any
            {
                std::unique_lock<std::mutex> hdc_lck(m_hdcMutex);
                if(!m_hostDeviceCopy.empty())
                {
                    hdcQueueType deviceData = std::move(m_hostDeviceCopy.front());
                    m_hostDeviceCopy.pop();
                    hdc_lck.unlock();

                    std::unique_ptr<HostDiscPT> hostData(new HostDiscPT(*(deviceData.first)));
                    hipEvent_t event;
                    hipEventCreate(&event);
                    hipEventRecord(event, 0);
                    m_ongoingTransfers.emplace(std::move(deviceData.first), std::move(hostData), deviceData.second,
                                               event);
                }
            }

            // handle finished memory transfers if any
            if(!m_ongoingTransfers.empty() && hipEventQuery(m_ongoingTransfers.front().event) != hipErrorNotReady)
            {
                OngoingTransfer transfer = std::move(m_ongoingTransfers.front());
                m_ongoingTransfers.pop();

                std::lock_guard<std::mutex> ddc_lck(m_ddcMutex);
                m_deviceDiskCopy.emplace(std::move(transfer.target),transfer.time);
            }

            // put data from cpu to files in memory
            {
                std::unique_lock<std::mutex> ddc_lck(m_ddcMutex);
                if(!m_deviceDiskCopy.empty())
                {
                    ddcQueueType hostData = std::move(m_deviceDiskCopy.front());
                    m_deviceDiskCopy.pop();
                    ddc_lck.unlock();

                    printTextFile(std::move(hostData));
                    m_numberJobs--;
                    logDEBUG("ResultStorageManager") << "Results stored for t= " << hostData.second;
                }
            }
        }
    }
}

ResultStorageManager::~ResultStorageManager()
{
    {
        std::lock_guard<std::mutex> lck(m_workerMutex);
        m_terminateWorker = true;
        m_workSignal.notify_one();
    }
    m_workerThread.join();
}

void ResultStorageManager::printTextFile(ResultStorageManager::ddcQueueType data)
{
    std::string filename = m_directory + m_prefix + m_startTime + "_" + mpu::toString(data.second)+".tsv";
    std::ofstream file(filename);

    if(!file.is_open())
    {
        logERROR("ResultStorageManager") << "Could not open output file: " << filename << " Make sure the path actually exists.";
        logFlush();
        throw std::runtime_error("Could not open output file.");
    }

    for(int i = 0; i < data.first->size(); ++i)
    {
        auto p = data.first->loadParticle<DiscPbases>(i);

        file << p.pos.x << "\t"
             << p.pos.y << "\t"
             << p.pos.z << "\t"
             << p.vel.x << "\t"
             << p.vel.y << "\t"
             << p.vel.z << "\t"
             << p.mass << "\t"
             << p.density << "\n";
    }

    if(file.fail())
    {
        logERROR("ResultStorageManager") << "Error writing output file: " << filename;
        logFlush();
        throw std::runtime_error("Could not write to output file.");
    }
}
