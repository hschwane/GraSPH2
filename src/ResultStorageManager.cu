/*
 * GraSPH2
 * ResultStorageManager.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the ResultStorageManager class, which saves simulation results into files.
 *
 * Copyright (c) 2018 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "ResultStorageManager.h"
//--------------------


// function definitions of the ResultStorageManager class
//-------------------------------------------------------------------

ResultStorageManager::ResultStorageManager(std::string directory, std::string prefix, int maxJobs)
    : m_directory(directory), m_prefix(prefix), m_terminateWorker(false), m_maxQueue(maxJobs),
    m_numberJobs(0), m_startTime(mpu::timestamp("%Y-%m-%d_%H:%M"))

{
    assert_critical(m_maxQueue>1, "ResultStorageManager", "Can't work with Maximum job number below 1.")
    m_workerThread = std::thread(&ResultStorageManager::worker, this);
}

void ResultStorageManager::worker()
{
    std::unique_lock<std::mutex> lck(m_workerMutex);
    HostDiscPT hostData;
    while(!m_terminateWorker)
    {
        // wait until there is work to do
        m_workSignal.wait(lck);

        while(m_numberJobs > 0)
        {
            // download data from gpu to cpu if any
            {
                std::unique_lock<std::mutex> hdc_lck(m_queueMutex);
                if(!m_deviceDiskCopy.empty())
                {
                    ddcQueueType deviceJob = std::move(m_deviceDiskCopy.front());
                    m_deviceDiskCopy.pop();
                    hdc_lck.unlock();

                    if(hostData.size() != deviceJob.first->size())
                    {
                        hostData = HostDiscPT(deviceJob.first->size());
                        hostData.pinMemory();
                    }
                    hostData = *deviceJob.first;
                    assert_cuda(hipGetLastError());

                    printTextFile(hostData,deviceJob.second);
                    m_numberJobs--;
                    logDEBUG("ResultStorageManager") << "Results stored for t= " << deviceJob.second;
                }
            }

            {
                std::unique_lock<std::mutex> hdc_lck(m_queueMutex);
                if(!m_hostDiskCopy.empty())
                {
                    hdcQueueType hostJob = std::move(m_hostDiskCopy.front());
                    m_hostDiskCopy.pop();
                    hdc_lck.unlock();

                    printTextFile(*hostJob.first, hostJob.second);
                    m_numberJobs--;
                    logDEBUG("ResultStorageManager") << "Results stored for t= " << hostJob.second;
                }
            }
        }
    }
}

ResultStorageManager::~ResultStorageManager()
{
    {
        std::lock_guard<std::mutex> lck(m_workerMutex);
        m_terminateWorker = true;
        m_workSignal.notify_one();
    }
    m_workerThread.join();
}

template<typename T>
void ResultStorageManager::attributePrinter::operator()(T v)
{
#ifndef __CUDA_ARCH__ // protection against calling from device code (mostly to shut up compiler warning)
    m_stream << v << "\t";
#endif
}

template<>
void ResultStorageManager::attributePrinter::operator()(f2_t v)
{
#ifndef __CUDA_ARCH__
    m_stream << v.x << "\t"
             << v.y << "\t";
#endif
}

template<>
void ResultStorageManager::attributePrinter::operator()(f3_t v)
{
#ifndef __CUDA_ARCH__
    m_stream << v.x << "\t"
             << v.y << "\t"
             << v.z << "\t";
#endif
}

template<>
void ResultStorageManager::attributePrinter::operator()(f4_t v)
{
#ifndef __CUDA_ARCH__
    m_stream << v.x << "\t"
             << v.y << "\t"
             << v.z << "\t"
             << v.w << "\t";
#endif
}

template<>
void ResultStorageManager::attributePrinter::operator()(m3_t v)
{
#ifndef __CUDA_ARCH__
    for(int i = 0; i < 9; ++i)
    {
        m_stream << v(i) << "\t";
    }
#endif
}

ResultStorageManager::attributePrinter::attributePrinter(std::ostream& s) : m_stream(s)
{
}

ResultStorageManager::attributePrinter::~attributePrinter()
{
    m_stream << "\n"; // particle finished, end the line
}

void ResultStorageManager::printTextFile(HostDiscPT& data, f1_t time)
{
    std::string filename = m_directory + m_prefix + m_startTime + "_" + mpu::toString(time)+".tsv";
    std::ofstream file(filename);

    if(!file.is_open())
    {
        logERROR("ResultStorageManager") << "Could not open output file: " << filename << " Make sure the path actually exists.";
        logFlush();
        throw std::runtime_error("Could not open output file.");
    }

    for(int i = 0; i < data.size(); ++i)
    {
        auto p = data.loadParticle(i);
        p.doForEachAttribute(attributePrinter(file));
    }

    if(file.fail())
    {
        logERROR("ResultStorageManager") << "Error writing output file: " << filename;
        logFlush();
        throw std::runtime_error("Could not write to output file.");
    }
}
