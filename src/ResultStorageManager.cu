#include "hip/hip_runtime.h"
/*
 * GraSPH2
 * ResultStorageManager.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the ResultStorageManager class, which saves simulation results into files.
 *
 * Copyright (c) 2018 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "ResultStorageManager.h"
//--------------------


// function definitions of the ResultStorageManager class
//-------------------------------------------------------------------

ResultStorageManager::ResultStorageManager(std::string directory, std::string prefix, int maxJobs)
    : m_directory(directory), m_prefix(prefix), m_terminateWorker(false), m_maxQueue(maxJobs),
    m_numberJobs(0), m_startTime(mpu::timestamp("%Y-%m-%d_%H:%M"))

{
    assert_critical(m_maxQueue>1, "ResultStorageManager", "Can't work with Maximum job number below 1.")
    m_workerThread = std::thread(&ResultStorageManager::worker, this);
}

void ResultStorageManager::worker()
{
    std::unique_lock<std::mutex> lck(m_workerMutex);
    HostDiscPT hostData;
    while(!m_terminateWorker)
    {
        // wait until there is work to do
        m_workSignal.wait(lck);

        while(m_numberJobs > 0)
        {
            // download data from gpu to cpu if any
            {
                std::unique_lock<std::mutex> hdc_lck(m_queueMutex);
                if(!m_deviceDiskCopy.empty())
                {
                    ddcQueueType deviceJob = std::move(m_deviceDiskCopy.front());
                    m_deviceDiskCopy.pop();
                    hdc_lck.unlock();

                    if(hostData.size() != deviceJob.first->size())
                    {
                        hostData = HostDiscPT(deviceJob.first->size());
                        hostData.pinMemory();
                    }
                    hostData = *deviceJob.first;
                    assert_cuda(hipGetLastError());

                    printHDF5File(hostData,deviceJob.second);
                    m_numberJobs--;
                    logDEBUG("ResultStorageManager") << "Results stored for t= " << deviceJob.second;
                }
            }

            {
                std::unique_lock<std::mutex> hdc_lck(m_queueMutex);
                if(!m_hostDiskCopy.empty())
                {
                    hdcQueueType hostJob = std::move(m_hostDiskCopy.front());
                    m_hostDiskCopy.pop();
                    hdc_lck.unlock();

                    printHDF5File(*hostJob.first, hostJob.second);
                    m_numberJobs--;
                    logDEBUG("ResultStorageManager") << "Results stored for t= " << hostJob.second;
                }
            }
        }
    }
}

ResultStorageManager::~ResultStorageManager()
{
    {
        std::lock_guard<std::mutex> lck(m_workerMutex);
        m_terminateWorker = true;
        m_workSignal.notify_one();
    }
    m_workerThread.join();
}
/*
template<typename T>
void ResultStorageManager::attributePrinter::operator()(T v)
{
#ifndef __CUDA_ARCH__ // protection against calling from device code (mostly to shut up compiler warning)
    m_stream << v << "\t";
#endif
}

template<>
void ResultStorageManager::attributePrinter::operator()(f2_t v)
{
#ifndef __CUDA_ARCH__
    m_stream << v.x << "\t"
             << v.y << "\t";
#endif
}

template<>
void ResultStorageManager::attributePrinter::operator()(f3_t v)
{
#ifndef __CUDA_ARCH__
    m_stream << v.x << "\t"
             << v.y << "\t"
             << v.z << "\t";
#endif
}

template<>
void ResultStorageManager::attributePrinter::operator()(f4_t v)
{
#ifndef __CUDA_ARCH__
    m_stream << v.x << "\t"
             << v.y << "\t"
             << v.z << "\t"
             << v.w << "\t";
#endif
}

template<>
void ResultStorageManager::attributePrinter::operator()(m3_t v)
{
#ifndef __CUDA_ARCH__
    for(int i = 0; i < 9; ++i)
    {
        m_stream << v(i) << "\t";
    }
#endif
}

ResultStorageManager::attributePrinter::attributePrinter(std::ostream& s) : m_stream(s)
{
}

ResultStorageManager::attributePrinter::~attributePrinter()
{
    m_stream << "\n"; // particle finished, end the line
}

void ResultStorageManager::printTextFile(HostDiscPT& data, f1_t time)
{
    std::ostringstream filename;
    filename << m_directory << m_prefix << m_startTime << "_" << std::fixed << std::setprecision(std::numeric_limits<f1_t>::digits10 + 1) << time << ".tsv";
    std::ofstream file(filename.str());
    file << std::fixed << std::setprecision(std::numeric_limits<f1_t>::digits10 + 1);

    if(!file.is_open())
    {
        logERROR("ResultStorageManager") << "Could not open output file: " << filename.str() << " Make sure the path actually exists.";
        logFlush();
        throw std::runtime_error("Could not open output file.");
    }

    for(int i = 0; i < data.size(); ++i)
    {
        auto p = data.loadParticle(i);
        p.doForEachAttribute(attributePrinter(file));
        std::vector<float> res;
        p.doForEachAttribute(attributeHDF5Printer(res));
    }

    if(file.fail())
    {
        logERROR("ResultStorageManager") << "Error writing output file: " << filename.str();
        logFlush();
        throw std::runtime_error("Could not write to output file.");
    }
}*/

template<typename A>
size_t getDim()
{return 0;}

template<>
size_t getDim<POS>()
{return 3;}

template<>
size_t getDim<VEL>()
{return 3;}

template<>
size_t getDim<MASS>()
{return 1;}

template<>
size_t getDim<DENSITY>()
{return 1;}

// Functions to convert particle data into atomic types

template<typename T>
void ResultStorageManager::attributeHDF5Printer::operator()(T v)
{
#ifndef __CUDA_ARCH__ // protection against calling from device code (mostly to shut up compiler warning)
    m_data.push_back(v);
#endif
}

template<>
void ResultStorageManager::attributeHDF5Printer::operator()(f2_t v)
{
#ifndef __CUDA_ARCH__
    m_data.push_back(v.x);
    m_data.push_back(v.y);
#endif
}

template<>
void ResultStorageManager::attributeHDF5Printer::operator()(f3_t v)
{
#ifndef __CUDA_ARCH__
    m_data.push_back(v.x);
    m_data.push_back(v.y);
    m_data.push_back(v.z);
#endif
}

template<>
void ResultStorageManager::attributeHDF5Printer::operator()(f4_t v)
{
#ifndef __CUDA_ARCH__
    m_data.push_back(v.x);
    m_data.push_back(v.y);
    m_data.push_back(v.z);
    m_data.push_back(v.w);
#endif
}

template<>
void ResultStorageManager::attributeHDF5Printer::operator()(m3_t v)
{
#ifndef __CUDA_ARCH__
    for(int i = 0; i < 9; ++i)
    {
        m_data.push_back(v(i));
    }
#endif
}

ResultStorageManager::attributeHDF5Printer::attributeHDF5Printer(std::vector<float>& s) : m_data(s)
{
}

ResultStorageManager::attributeHDF5Printer::~attributeHDF5Printer()
{
}

template <typename A>
void writeAttributeDataset(const HostDiscPT& data, HighFive::File& file)
{
    using namespace HighFive;
    try
    {
        //Create DataSpace for DataSet (min size and max size);
        DataSpace dspace = HighFive::DataSpace({data.size(), getDim<A>()});
        // Create a new Dataset
        DataSet dset = file.createDataSet(std::string(A::debugName()), dspace, AtomicType<float>());

        //One long float vector, in which the position/ density/ mass / vel is stored
        std::vector<float> res;

        // create dataset ... A::debugName();
        for (int i = 0; i < data.size(); ++i)
        {
            auto p = data.loadParticle<A>(i);
            std::vector<float> res;
            p.doForEachAttribute(ResultStorageManager::attributeHDF5Printer(res));
            dset.select({size_t(i),0},{1,res.size()}).write(res);
        }
    }
    catch(const Exception& err)
    {
        std::cerr << err.what() << std::endl;
    }
}

template<typename ...Args>
struct writeAllParticles
{
    void operator()(const HostDiscPT &data, HighFive::File& file)
    {
        int t[] = {0, ((void) (writeAttributeDataset<Args>(data, file)), 1)...};
        (void) t[0]; // silence compiler warning about t being unused
    }
};

void ResultStorageManager::printHDF5File(HostDiscPT& data, f1_t time)
{
    std::ostringstream filename;
    filename << m_directory << m_prefix << m_startTime << "_" << std::fixed << std::setprecision(std::numeric_limits<f1_t>::digits10 + 1) << time << ".h5";

    //Create HDF5 File and DataSet which stores the result of one time step (all attributes of all particles at this timestep)
    HighFive::File file(filename.str(), HighFive::File::ReadWrite | HighFive::File::Create | HighFive::File::Truncate);

    mpu::instantiate_from_tuple_t<writeAllParticles, HostDiscPT::particleType::attributes> myWriteFunction;
    myWriteFunction(data, file);
}
