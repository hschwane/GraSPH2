#include "hip/hip_runtime.h"
/*
 * mpUtils
 * main.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail: hendrik.schwanekamp@gmx.net
 *
 * mpUtils = my personal Utillities
 * A utility library for my personal c++ projects
 *
 * Copyright 2016 Hendrik Schwanekamp
 *
 */

#include <mpUtils.h>
#include <Cuda/cudaUtils.h>

#include "Particles.h"
#include "frontends/frontendInterface.h"

__global__ void test(Particles* from, Particles* to)
{
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
//    int stride = blockDim.x * gridDim.x;

    SharedParticles<100,SHARED_POSM> sp;
    sp.copyFromGlobal(threadIdx.x, index, *from);

    auto p = sp.loadParticle<POSM>(index);

    to->storeParticle(p,index);
}

int main()
{

    mpu::Log myLog( mpu::LogLvl::ALL, mpu::ConsoleSink());


    logINFO("pfSPH") << "Welcome to planetformSPH!";
    fnd::initializeFrontend();

    bool simShouldRun = false;
    fnd::setPauseHandler([&simShouldRun](bool pause){simShouldRun = !pause;});

    Particles* pb1 = new Particles(100);
    Particles* pb2 = new Particles(100);

    Particle<M,VEL> p;
    p.vel = {12,56,85};
    p.mass = 10.0f;
    pb1->storeParticle(p,10);

    assert_cuda(hipDeviceSynchronize());
    pb1->copyToDevice();
    assert_cuda(hipDeviceSynchronize());

    test<<<1,100>>>(pb1,pb2);

    assert_cuda(hipDeviceSynchronize());
    pb2->copyFromDevice();
    assert_cuda(hipDeviceSynchronize());
    p = pb2->loadParticle<M, VEL>(10);

    logINFO("test") << pb2->loadParticle<M>(10).mass;

    mpu::DeltaTimer dt;
    while(fnd::handleFrontend(dt.getDeltaTime()))
    {
        if(simShouldRun)
        {
            // run simulation here
        }
    }

    return 0;
}
