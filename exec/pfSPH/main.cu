#include "hip/hip_runtime.h"
/*
 * mpUtils
 * main.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail: hendrik.schwanekamp@gmx.net
 *
 * mpUtils = my personal Utillities
 * A utility library for my personal c++ projects
 *
 * Copyright 2016 Hendrik Schwanekamp
 *
 */

#include <mpUtils.h>
#include <cuda_gl_interop.h>
#include <thrust/random.h>

#include "particles/Particles.h"
#include "frontends/frontendInterface.h"
#include <Cuda/cudaUtils.h>
#include <crt/math_functions.hpp>
#include "particles/algorithms.h"

constexpr int BLOCK_SIZE = 256;
constexpr int PARTICLES = 1<<15;

int NUM_BLOCKS = (PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE;

__global__ void generate2DNBSystem(Particles<DEV_POSM,DEV_VEL,DEV_ACC> particles)
{
    INIT_EACH(particles, MPU_COMMA_LIST(POS,MASS,VEL),
    {
        thrust::random::default_random_engine rng;
        rng.discard(i);
        thrust::random::uniform_real_distribution<float> dist(-1.0f,1.0f);

        pi.pos.x = dist(rng);
        pi.pos.y = dist(rng);
        pi.pos.z = 0.0f;
        pi.mass = 1.0f/particles.size();

        pi.vel = cross(pi.pos,{0.0f,0.0f, 0.75f});
    });
}

__global__ void nbodyForces(Particles<DEV_POSM,DEV_VEL,DEV_ACC> particles, f1_t eps2, const int numTiles)
{
    DO_FOR_EACH_PAIR_SM( BLOCK_SIZE, particles, SHARED_POSM, MPU_COMMA_LIST(POS,MASS,VEL,ACC),
                        MPU_COMMA_LIST(POS,MASS,VEL), MPU_COMMA_LIST(ACC), MPU_COMMA_LIST(POS, MASS),
    {
        f3_t r = pi.pos - pj.pos;
        f1_t distSqr = dot(r, r) + eps2;

        f1_t invDist = rsqrt(distSqr);
        f1_t invDistCube = invDist * invDist * invDist;
        pi.acc -= r * pj.mass * invDistCube;
    },
    {
         pi.acc -= pi.vel * 0.01;
    })
}

__global__ void integrateLeapfrog(Particles<DEV_POSM,DEV_VEL,DEV_ACC> particles, f1_t dt, bool not_first_step)
{
    DO_FOR_EACH(particles, MPU_COMMA_LIST(POS,VEL,ACC), MPU_COMMA_LIST(POS,VEL,ACC), MPU_COMMA_LIST(POS,VEL),
    {
        //   calculate velocity a_t
        pi.vel = pi.vel + pi.acc * (dt * 0.5f);

        // we could now change delta t here

        // calculate velocity a_t+1/2
        pi.vel = pi.vel + pi.acc * (dt * 0.5f) * not_first_step;

        // calculate position r_t+1
        pi.pos = pi.pos + pi.vel * dt;
    })
}

int main()
{
    mpu::Log myLog( mpu::LogLvl::ALL, mpu::ConsoleSink());

    logINFO("pfSPH") << "Welcome to planetformSPH!";
    assert_cuda(hipSetDevice(0));

    // set up frontend
    fnd::initializeFrontend();
    bool simShouldRun = true;
    fnd::setPauseHandler([&simShouldRun](bool pause){simShouldRun = !pause;});

    // generate some particles
    Particles<DEV_POSM,DEV_VEL,DEV_ACC> pb(PARTICLES);

    // register position and velocity buffer with cuda
#if defined(FRONTEND_OPENGL)
    pb.registerGLGraphicsResource<DEV_POSM>(fnd::getPositionBuffer(pb.size()));
    pb.registerGLGraphicsResource<DEV_VEL>(fnd::getVelocityBuffer(pb.size()));
    pb.mapGraphicsResource();
#endif

    generate2DNBSystem<<<NUM_BLOCKS,BLOCK_SIZE>>>(pb.createDeviceCopy());
    assert_cuda(hipGetLastError());
    assert_cuda(hipDeviceSynchronize());

    nbodyForces<<<NUM_BLOCKS,BLOCK_SIZE>>>(pb.createDeviceCopy(),0.0001f, PARTICLES/ BLOCK_SIZE);
    assert_cuda(hipGetLastError());
    integrateLeapfrog<<<NUM_BLOCKS,BLOCK_SIZE>>>(pb.createDeviceCopy(),0.001f,false);
    assert_cuda(hipGetLastError());

    pb.unmapGraphicsResource(); // used for frontend stuff
    mpu::DeltaTimer dt;
    while(fnd::handleFrontend(dt.getDeltaTime()))
    {
        if(simShouldRun)
        {
            pb.mapGraphicsResource(); // used for frontend stuff

            nbodyForces<<<NUM_BLOCKS,BLOCK_SIZE>>>(pb.createDeviceCopy(),0.0001f, PARTICLES/ BLOCK_SIZE);
            assert_cuda(hipGetLastError());
            integrateLeapfrog<<<NUM_BLOCKS,BLOCK_SIZE>>>(pb.createDeviceCopy(),0.001f,true);
            assert_cuda(hipGetLastError());

            pb.unmapGraphicsResource(); // used for frontend stuff
        }
    }

    return 0;
}