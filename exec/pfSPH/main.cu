#include "hip/hip_runtime.h"
/*
 * mpUtils
 * main.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail: hendrik.schwanekamp@gmx.net
 *
 * mpUtils = my personal Utillities
 * A utility library for my personal c++ projects
 *
 * Copyright 2016 Hendrik Schwanekamp
 *
 */

#include <thrust/random.h>
#include <mpUtils.h>
#include <mpCuda.h>
#include <cuda_gl_interop.h>

#include "frontends/frontendInterface.h"
#include "particles/Particles.h"
#include "particles/algorithms.h"
#include "sph/kernel.h"
#include "sph/eos.h"

constexpr int BLOCK_SIZE = 256;
constexpr int PARTICLES = 1<<13;
constexpr f1_t H = 0.022;

constexpr f1_t alpha = 1;
constexpr f1_t rho0 = 0.5;
constexpr f1_t BULK = 10;
constexpr f1_t dBULKdP = 1;
constexpr f1_t shear = 6;
const f1_t SOUNDSPEED = sqrt(BULK / rho0);

int NUM_BLOCKS = (PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE;

using DeviceParticlesType = Particles<DEV_POSM,DEV_VEL,DEV_ACC,DEV_DENSITY,DEV_DENSITY_DT,DEV_DSTRESS,DEV_DSTRESS_DT>;

__global__ void generate2DRings(DeviceParticlesType particles)
{
    const float R = 0.5;
    const float r = 0.4;
    const float a = M_PI * (R*R-r*r);
    const float ringMass = rho0 * a;
    const int ringSize = particles.size()/2;
    const float particleMass = ringMass/ringSize;

    const float xOffA = -0.6;
    const float xOffB = 0.6;
    const float speed = .25;


    INIT_EACH(particles, MPU_COMMA_LIST(POS,MASS,VEL,DENSITY),
    {
        thrust::random::default_random_engine rng;
        rng.discard(i);
        thrust::random::uniform_real_distribution<float> dist(-1.0f,1.0f);

        while(length(pi.pos) > R || length(pi.pos) < r)
        {
            pi.pos.x = dist(rng);
            rng.discard(particles.size());
            pi.pos.y = dist(rng);
            rng.discard(particles.size());
        }

        if(i < ringSize)
        {
            pi.pos.x += xOffA;
            pi.vel.x = speed;
        }
        else
        {
            pi.pos.x += xOffB;
            pi.vel.x = -speed;
        }

        pi.mass = particleMass;
        pi.density = rho0;
    });
}

__global__ void generateSquares(DeviceParticlesType particles)
{
    INIT_EACH(particles, MPU_COMMA_LIST(POS,MASS,VEL,DENSITY),
    {
        float spacing = H/2;
        int squareSize = particles.size()/2;
        int sideres = sqrt(float(squareSize));
        float side = (sideres-1) * spacing;

        const float a = side*side;
        const float squareMass = rho0 * a;
        const float particleMass = squareMass/squareSize;

        const float speed = .25;
        const float seperation = 1;

        if(i < squareSize)
        {
            pi.pos.x = -side / 2 + (i%sideres) *spacing;
            pi.pos.y = -side / 2 + (i/sideres) *spacing;
            pi.pos.x -= seperation/2;
            pi.pos.y -= seperation/4;
            pi.vel.x = speed;
        }
        else
        {
            pi.pos.x = -side / 2 + ((i-squareSize)%sideres) *spacing;
            pi.pos.y = -side / 2 + ((i-squareSize)/sideres) *spacing;
            pi.pos.x += seperation/2;
            pi.pos.y += seperation/4;
            pi.vel.x = -speed;
        }

        pi.mass = particleMass;
        pi.density = rho0;
    })
}

__device__ f1_t artificialViscosity(f1_t alpha, f1_t density_i, f1_t density_j, const f3_t& vij,  const f3_t& rij, f1_t r, f1_t ci, f1_t cj)
{
    const f1_t wij = dot(rij, vij) /r;
    f1_t II = 0;
    if(wij < 0)
    {
        const f1_t vsig = f1_t(ci+cj - 3.0*wij);
        const f1_t rhoij = (density_i + density_j)*f1_t(0.5);
        II = -0.5f * alpha * wij * vsig / rhoij;
    }
    return II;
}

__global__ void computeDerivatives(DeviceParticlesType particles, f1_t speedOfSound)
{
    DO_FOR_EACH_PAIR_SM( BLOCK_SIZE, particles, MPU_COMMA_LIST(SHARED_POSM,SHARED_VEL,SHARED_DENSITY,SHARED_DSTRESS),
            MPU_COMMA_LIST(POS,MASS,VEL,ACC,DENSITY,DENSITY_DT,DSTRESS,DSTRESS_DT),
            MPU_COMMA_LIST(POS,MASS,VEL,DENSITY,DSTRESS), MPU_COMMA_LIST(ACC,DENSITY_DT,DSTRESS_DT),
            MPU_COMMA_LIST(POS,MASS,VEL,DENSITY,DSTRESS),

    int numPartners=0;
    m3_t sigOverRho_i; // stress over density square used for acceleration
    m3_t edot(0); // strain rate tensor (edot)
    m3_t rdot(0); // rotation rate tensor
    f1_t vdiv{0}; // velocity divergence
    {
        sigOverRho_i = pi.dstress;
        f1_t pres_i = eos::murnaghan( pi.density, rho0, BULK, dBULKdP);
        sigOverRho_i[0][0] = (sigOverRho_i[0][0] - pres_i) / (pi.density*pi.density);
        sigOverRho_i[1][1] = (sigOverRho_i[1][1] - pres_i) / (pi.density*pi.density);
        sigOverRho_i[2][2] = (sigOverRho_i[2][2] - pres_i) / (pi.density*pi.density);
    }
    ,
    {
        const f3_t rij = pi.pos-pj.pos;
        const f1_t r2 = dot(rij,rij);
        f1_t r = sqrt(r2);
        if(r>0 && r <= H)
        {
            numPartners++;
            // get the kernel gradient
            const f1_t dw = kernel::dWspline<Dim::two>(r,H);
            const f3_t gradw = (dw/r) * rij;

            // artificial viscosity
            const f3_t vij = pi.vel-pj.vel;
            pi.acc -= pj.mass * artificialViscosity(alpha,pi.density,pj.density,vij,rij,r,speedOfSound,speedOfSound) * gradw;

            // pressure
            m3_t sigOverRho_j = pj.dstress;
            f1_t pres_j = eos::murnaghan( pj.density, rho0, BULK, dBULKdP);
            sigOverRho_j[0][0] = (sigOverRho_j[0][0] - pres_j) / (pj.density*pj.density);
            sigOverRho_j[1][1] = (sigOverRho_j[1][1] - pres_j) / (pj.density*pj.density);
            sigOverRho_j[2][2] = (sigOverRho_j[2][2] - pres_j) / (pj.density*pj.density);

            // acceleration
            pi.acc.x += pj.mass * ((sigOverRho_i[0][0]+sigOverRho_j[0][0])*gradw.x + (sigOverRho_i[0][1]+sigOverRho_j[0][1])*gradw.y + (sigOverRho_i[0][2]+sigOverRho_j[0][2])*gradw.z);
            pi.acc.y += pj.mass * ((sigOverRho_i[1][0]+sigOverRho_j[1][0])*gradw.x + (sigOverRho_i[1][1]+sigOverRho_j[1][1])*gradw.y + (sigOverRho_i[1][2]+sigOverRho_j[1][2])*gradw.z);
            pi.acc.z += pj.mass * ((sigOverRho_i[2][0]+sigOverRho_j[2][0])*gradw.x + (sigOverRho_i[2][1]+sigOverRho_j[2][1])*gradw.y + (sigOverRho_i[2][2]+sigOverRho_j[2][2])*gradw.z);

            // strain rate tensor (edot) and rotation rate tensor (rdot)
            f1_t tmp= -0.5f * pj.mass/pi.density;
            edot[0][0] += tmp*(vij.x*gradw.x + vij.x*gradw.x);
            edot[0][1] += tmp*(vij.x*gradw.y + vij.y*gradw.x);
            edot[0][2] += tmp*(vij.x*gradw.z + vij.z*gradw.x);
            edot[1][0] += tmp*(vij.y*gradw.x + vij.x*gradw.y);
            edot[1][1] += tmp*(vij.y*gradw.y + vij.y*gradw.y);
            edot[1][2] += tmp*(vij.y*gradw.z + vij.z*gradw.y);
            edot[2][0] += tmp*(vij.z*gradw.x + vij.x*gradw.z);
            edot[2][1] += tmp*(vij.z*gradw.y + vij.y*gradw.z);
            edot[2][2] += tmp*(vij.z*gradw.z + vij.z*gradw.z);

            rdot[0][0] += tmp*(vij.x*gradw.x - vij.x*gradw.x);
            rdot[0][1] += tmp*(vij.x*gradw.y - vij.y*gradw.x);
            rdot[0][2] += tmp*(vij.x*gradw.z - vij.z*gradw.x);
            rdot[1][0] += tmp*(vij.y*gradw.x - vij.x*gradw.y);
            rdot[1][1] += tmp*(vij.y*gradw.y - vij.y*gradw.y);
            rdot[1][2] += tmp*(vij.y*gradw.z - vij.z*gradw.y);
            rdot[2][0] += tmp*(vij.z*gradw.x - vij.x*gradw.z);
            rdot[2][1] += tmp*(vij.z*gradw.y - vij.y*gradw.z);
            rdot[2][2] += tmp*(vij.z*gradw.z - vij.z*gradw.z);


            // density time derivative
            vdiv += (pj.mass/pj.density) * dot(vij,gradw);
        }
    },
    {
//        printf("%i\n",numPartners);
        // density time derivative
        pi.density_dt = pi.density * vdiv;

        // deviatoric stress time derivative
        for(int d = 0; d < 3; ++d)
            for(int e = 0; e < 3; ++e)
            {
                pi.dstress_dt[d][e] += 2*shear*edot[d][e];
                for(int f=0; f<3;f++)
                {
                    if(d==e)
                    {
                        pi.dstress_dt[d][e] += 2*shear*edot[f][f] / 3.0f;
                    }
                    pi.dstress_dt[d][e] += pi.dstress[d][f]*rdot[e][f];
                    pi.dstress_dt[d][e] += pi.dstress[e][f]*rdot[d][f];
                }

            }
    })
}

__global__ void integrate(DeviceParticlesType particles, f1_t dt)
{
    DO_FOR_EACH(particles, MPU_COMMA_LIST(POS,VEL,ACC,DENSITY,DENSITY_DT,DSTRESS,DSTRESS_DT),
            MPU_COMMA_LIST(POS,VEL,ACC,DENSITY,DENSITY_DT,DSTRESS,DSTRESS_DT),
            MPU_COMMA_LIST(POS,VEL,DENSITY,DSTRESS),
    {
        // eqn of motion
        pi.vel += pi.acc * dt;
        pi.pos += pi.vel * dt;

        // density
        pi.density += pi.density_dt * dt;

        // deviatoric stress
        pi.dstress += pi.dstress_dt * dt;
    })
}

__global__ void generate2DNBSystem(DeviceParticlesType particles)
{
    INIT_EACH(particles, MPU_COMMA_LIST(POS,MASS,VEL),
              {
                  thrust::random::default_random_engine rng;
                  rng.discard(i);
                  thrust::random::uniform_real_distribution<float> dist(-1.0f,1.0f);

                  pi.pos.x = dist(rng);
                  pi.pos.y = dist(rng);
                  pi.pos.z = 0.0f;
                  pi.mass = 1.0f/particles.size();

                  pi.vel = cross(pi.pos,{0.0f,0.0f, 0.75f});
              });
}

__global__ void nbodyForces(DeviceParticlesType particles, f1_t eps2)
{
    DO_FOR_EACH_PAIR_SM( BLOCK_SIZE, particles, SHARED_POSM, MPU_COMMA_LIST(POS,MASS,VEL,ACC),
                         MPU_COMMA_LIST(POS,MASS,VEL), MPU_COMMA_LIST(ACC), MPU_COMMA_LIST(POS, MASS), {},
    {
        f3_t r = pi.pos - pj.pos;
        f1_t distSqr = dot(r, r) + eps2;

        f1_t invDist = rsqrt(distSqr);
        f1_t invDistCube = invDist * invDist * invDist;
        pi.acc -= r * pj.mass * invDistCube;
    },
    {
        pi.acc -= pi.vel * 0.01;
    })
}

__global__ void integrateLeapfrog(DeviceParticlesType particles, f1_t dt, bool not_first_step)
{
    DO_FOR_EACH(particles, MPU_COMMA_LIST(POS,VEL,ACC), MPU_COMMA_LIST(POS,VEL,ACC), MPU_COMMA_LIST(POS,VEL),
    {
        //   calculate velocity a_t
        pi.vel = pi.vel + pi.acc * (dt * 0.5f);

        // we could now change delta t here

        // calculate velocity a_t+1/2
        pi.vel = pi.vel + pi.acc * (dt * 0.5f) * not_first_step;

        // calculate position r_t+1
        pi.pos = pi.pos + pi.vel * dt;
    })
}

int main()
{
    mpu::Log myLog( mpu::LogLvl::ALL, mpu::ConsoleSink());

    logINFO("pfSPH") << "Welcome to planetformSPH!";
    assert_cuda(hipSetDevice(0));

    // set up frontend
    fnd::initializeFrontend();
    bool simShouldRun = false;
    fnd::setPauseHandler([&simShouldRun](bool pause){simShouldRun = !pause;});

    // generate some particles
    DeviceParticlesType pb(PARTICLES);
    pb.initialize();

    // register position and velocity buffer with cuda
#if defined(FRONTEND_OPENGL)
    pb.registerGLGraphicsResource<DEV_POSM>(fnd::getPositionBuffer(pb.size()));
    pb.registerGLGraphicsResource<DEV_VEL>(fnd::getVelocityBuffer(pb.size()));
    pb.mapGraphicsResource();
#endif

    generateSquares<<<NUM_BLOCKS,BLOCK_SIZE>>>(pb.createDeviceCopy());
    assert_cuda(hipGetLastError());
    assert_cuda(hipDeviceSynchronize());

    pb.unmapGraphicsResource(); // used for frontend stuff
    while(fnd::handleFrontend())
    {
        if(simShouldRun)
        {
            pb.mapGraphicsResource(); // used for frontend stuff

            computeDerivatives<<<NUM_BLOCKS,BLOCK_SIZE>>>(pb.createDeviceCopy(),SOUNDSPEED);
            assert_cuda(hipGetLastError());
            integrate<<<NUM_BLOCKS,BLOCK_SIZE>>>(pb.createDeviceCopy(),0.0003f);
            assert_cuda(hipGetLastError());

            pb.unmapGraphicsResource(); // used for frontend stuff
        }
    }

    return 0;
}
