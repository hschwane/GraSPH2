#include "hip/hip_runtime.h"
/*
 * mpUtils
 * main.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail: hendrik.schwanekamp@gmx.net
 *
 * mpUtils = my personal Utillities
 * A utility library for my personal c++ projects
 *
 * Copyright 2016 Hendrik Schwanekamp
 *
 */

#include <mpUtils.h>
#include <cuda_gl_interop.h>
#include <thrust/random.h>

#include "particles/Particles.h"
#include "frontends/frontendInterface.h"
#include <Cuda/cudaUtils.h>
#include <crt/math_functions.hpp>



constexpr int BLOCK_SIZE = 256;
constexpr int PARTICLES = 1<<15;

int NUM_BLOCKS = (PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE;

__global__ void generate2DNBSystem(Particles<DEV_POSM,DEV_VEL,DEV_ACC> particles)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx>=particles.size())
    {
        printf("wrong dispatch parameters for particle count!");
        return;
    }

    thrust::random::default_random_engine rng;
    rng.discard(idx);
    thrust::random::uniform_real_distribution<float> dist(-1.0f,1.0f);

    Particle<POS,MASS,VEL,ACC> p;

    p.pos.x = dist(rng);
    p.pos.y = dist(rng);
    p.pos.z = 0.0f;
    p.mass = 1.0f/particles.size();

    p.vel = cross(p.pos,{0.0f,0.0f, 0.75f});

    particles.storeParticle(idx,p);
}

__global__ void nbodyForces(Particles<DEV_POSM,DEV_VEL,DEV_ACC> particles, f1_t eps2, const int numTiles)
{
    SharedParticles<BLOCK_SIZE,SHARED_POSM> shared;

    const unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

    Particle<POS,MASS,VEL,ACC> pi = particles.loadParticle<POS,VEL,MASS>(idx);

    for (int tile = 0; tile < numTiles; tile++)
    {
        const auto p = particles.loadParticle<POS,MASS>(tile*blockDim.x+threadIdx.x);
        shared.storeParticle(threadIdx.x,p);

        __syncthreads();

        for(int j = 0; j<blockDim.x;j++)
        {
            auto pj = shared.loadParticle<POS,MASS>(j);
            f3_t r = pi.pos-pj.pos;
            f1_t distSqr = dot(r,r) + eps2;

            f1_t invDist = rsqrt(distSqr);
            f1_t invDistCube =  invDist * invDist * invDist;
            pi.acc -= r * pj.mass * invDistCube;

        }
        __syncthreads();
    }
    pi.acc -= pi.vel * 0.1;
    particles.storeParticle(idx,Particle<ACC>(pi));
}


__global__ void integrateLeapfrog(Particles<DEV_POSM,DEV_VEL,DEV_ACC> particles, f1_t dt, bool not_first_step)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx>=particles.size())
    {
        printf("wrong dispatch parameters for particle count!");
        return;
    }

    auto pi = particles.loadParticle<POS,MASS,VEL,ACC>(idx);

    //   calculate velocity a_t
    pi.vel  = pi.vel + pi.acc * (dt*0.5f);

    // we could now change delta t here

    // calculate velocity a_t+1/2
    pi.vel = pi.vel + pi.acc * (dt*0.5f) * not_first_step;

    // calculate position r_t+1
    pi.pos = pi.pos + pi.vel * dt;

    particles.storeParticle(idx,pi);
}

int main()
{

    mpu::Log myLog( mpu::LogLvl::ALL, mpu::ConsoleSink());

    logINFO("pfSPH") << "Welcome to planetformSPH!";
    assert_cuda(hipSetDevice(0));

    // set up frontend
    fnd::initializeFrontend();
    bool simShouldRun = true;
    fnd::setPauseHandler([&simShouldRun](bool pause){simShouldRun = !pause;});

    // generate some particles
//    Particles<DEV_POSM,DEV_VEL,DEV_ACC> pb(PARTICLES);

    DEV_POSM dp(5);

    // register position and velocity buffer with cuda
//#if defined(FRONTEND_OPENGL)
//    static_cast<DEV_POSM>(pb).registerGLGraphicsResource(fnd::getPositionBuffer(pb.size()));
//    static_cast<DEV_VEL>(pb).registerGLGraphicsResource(fnd::getVelocityBuffer(pb.size()));
//    pb.mapGraphicsResource();
//#endif

    dp.registerGLGraphicsResource(fnd::getPositionBuffer(5));
    dp.mapGraphicsResource();

    Particle<POS,MASS,VEL> p;
    HOST_POSM hp(5);
    p.pos = {0.5,0.5,0};
    p.vel = {1,0,0};
    hp.storeParticle(2,p);

    p.pos = {-0.5,0.5,0};
    p.vel = {1,0,0};
    hp.storeParticle(1,p);

    p.pos = {0.5,-0.5,0};
    p.vel = {1,0,0};
    hp.storeParticle(0,p);

    p.pos = {-0.5,-0.5,0};
    p.vel = {1,0,0};
    hp.storeParticle(3,p);

    dp = hp;
    hp=dp;

    hp.loadParticle(2,p);
    std::cout << p.pos.x << " " << p.pos.y << " " << p.pos.z <<std::endl;

    dp.unmapGraphicsResource();

//    generate2DNBSystem<<<NUM_BLOCKS,BLOCK_SIZE>>>(pb.createDeviceCopy());
    assert_cuda(hipGetLastError());
    assert_cuda(hipDeviceSynchronize());

//    nbodyForces<<<NUM_BLOCKS,BLOCK_SIZE>>>(pb.createDeviceCopy(),0.01f, PARTICLES/ BLOCK_SIZE);
//    assert_cuda(hipGetLastError());
//    integrateLeapfrog<<<NUM_BLOCKS,BLOCK_SIZE>>>(pb.createDeviceCopy(),0.005f,false);
//    assert_cuda(hipGetLastError());

//    pb.unmapGraphicsResource(); // used for frontend stuff
    mpu::DeltaTimer dt;
    while(fnd::handleFrontend(dt.getDeltaTime()))
    {
        if(simShouldRun)
        {
//            pb.mapGraphicsResource(); // used for frontend stuff

//            nbodyForces<<<NUM_BLOCKS,BLOCK_SIZE>>>(pb.createDeviceCopy(),0.00001f, PARTICLES/ BLOCK_SIZE);
//            assert_cuda(hipGetLastError());
//            integrateLeapfrog<<<NUM_BLOCKS,BLOCK_SIZE>>>(pb.createDeviceCopy(),0.0025f,true);
//            assert_cuda(hipGetLastError());

//            pb.unmapGraphicsResource(); // used for frontend stuff
        }
    }

    return 0;
}